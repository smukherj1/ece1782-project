#include "hip/hip_runtime.h"
#include "gpu_sort.h"

#include <stdio.h>

#define checkCudaOK(val) {\
    if(val != hipSuccess) {\
        fprintf(stderr, "Cuda check failed at %s:%d '%s'\n", __FILE__, __LINE__, #val); \
        fprintf(stderr, "%s\n", hipGetErrorString(val));\
        exit(EXIT_FAILURE);\
    }\
}

__global__ void bitonic_sort_kernel()
{
	printf("Hello World from thread %d in the GPU!\n", threadIdx.x);
}


void gpu_sort()
{
	bitonic_sort_kernel<<<1, 1>>> ();
	checkCudaOK(hipDeviceSynchronize());
}